#include "hip/hip_runtime.h"
﻿
/* Autores:
 *	Daniel López Moreno
 *	Diego-Edgar Gracia Peña
 * Enunciado:
 *	Juego de 16384
 *		Versión en CUDA del Juego 2048
 *
 * Sin Bloques ni Memoria Compartida
 */

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <conio.h>
#include <stdlib.h>
#include <Windows.h>
#include <time.h>

#include <fstream>
#include <iostream>
#include <string>
	using namespace std;

const int WS = 6;
int BAJO[] = { 2, 4, 8 };
int ALTO[] = { 2, 4 };
int VIDAS = 5;
char MODO[] = "-m";
char FICHERO[] = "16384.sav";

__device__ int TILE_WIDTH_M = 0;
__device__ int TILE_WIDTH_N = 0;


//	Ejemplo de como quedaría la matriz

//		-	-	N	-	-
//	|	00	01	02	03	04
//	M	10	11	12	13	14
//	|	20	21	22	23	24

//M es el eje Y
//N es el eje X

//------------------------------------------------------------------- Device ------------------------------------------------------------------------

//	Inicializador de la matriz de juego
//	-	*m Matriz en forma vectorial con la que se trabaja, WidthM y WidthN su tamaño de columna y fila
__global__ void Inicializador(int *m, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	if (id_fil < WidthM && id_col < WidthN) {//Comprobación de que el hilo esté dentro de los límites
		m[id_fil*WidthN + id_col] = 0;
	}
}

//	Inicializador de matrices booleanas
//	-	*b Matriz vectorial de booleanos, WidthM y WidthN dimensiones de columna y fila
//	-	set el valor booleano a introducir
__global__ void iniBool(bool *b, int WidthM, int WidthN, bool set) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	if (id_fil < WidthM && id_col < WidthN) {//Comprobación de que el hilo esté dentro de los límites

		b[id_fil*WidthN + id_col] = set;//damos al elemento correspondiente el valor indicado

	}
}

//---------------------- Dch -------------------------

//	Cada hilo busca una pareja para su elemento correspondiente, y si es viable, realiza la suma
//	-	Cada hilo recorre la matriz hacia la derecha buscando fichas como la suya viables para la suma
//	-	para ello, cuenta cuantas coincidencias hay, si el numero es congruente con 0 mod 2,
//	-	no se realizará ninguna acción por parte del hilo, si es congruente con 1 mod 2,
//	-	se multiplica por 2 el primer coincidente y se borra la ficha del hilo.
//	-	Las coincidencias deben de ser inmediatas, solo permitiendose el 0 entre las fichas (0 == vacio)
//	-	-	La puntuación se recoge en la matriz p
__global__ void SumaDch(int *m, int *p, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, c = 0, aux, i;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		//si la ficha está vacia, el hilo no buscará
		if (ficha != 0) {
			//Se realiza la busqueda hacia la dch
			for (i = id_col + 1; i < WidthN; i++) {
				aux = m[id_fil*WidthN + i];

				if (aux == ficha) c++;//contamos las coincidencias
				else if (aux != 0) i = WidthN;//No podemos emparejar saltandonos fichas
			}

			//	Si el numero de coincidencias es congruente con 1 mod 2
			//	se busca la primera coincidencia, se multiplica por 2 y se borra la ficha 
			//	Si fuese congruente con 0 mod 2, no debe acceder al for
			if ((c % 2) == 0) p[id_fil*WidthN + id_col] = 0;//	Si no opera, puntuación 0
			for (i = id_col + 1; i < WidthN && (c % 2) == 1; i++) {
				aux = m[id_fil*WidthN + i];
				if (aux == ficha) {
					m[id_fil*WidthN + i] = ficha * 2;
					m[id_fil*WidthN + id_col] = 0;
					p[id_fil*WidthN + id_col] = ficha * 2;//	Grabamos la puntuación obtenido con la suma
					c--;//Para que el bucle for termine
				}

				//	(Aclaración) Si estamos entrando en este bucle for,
				//		significa que se ha encontrado una pareja viable anteriormente
				//		por lo que no se filtra si se opera con una ficha no válida
			}
		}
		else p[id_fil*WidthN + id_col] = 0;
	}
}

//	Ejecución de Movimiento a la Derecha de las piezas
//	-	Cada hilo toma su ficha (si es distinta de 0) y busca espacios en blanco a su derecha
//	-	Cuando no encuentra más huecos en la matriz, intercambia su ficha con la del último hueco hallado
//	-	al ser 0, intercambia con una vacía, si no hubiese huecos a su derecha, la intercambia consigo mismo
//	-	-	Esta función debe ser llamada hasta que no devuelva ningún cambio en la Matriz de Juego
__global__ void exMovDch(int *m, bool *b, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, id_aux = id_col;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		if (ficha != 0) {//si es 0, no hay que hacer ningún movimiento
			for (int i = id_col + 1; i < WidthN; i++) {
				if (m[id_fil*WidthN + i] == 0) id_aux = i;//se va buscando huecos vacios
				else i = WidthN;//hasta toparse con otra ficha, entonces paramos la búsqueda
			}

			//Intercambiamos las fichas, aunque no se haya encontrado ningún hueco
			m[id_fil*WidthN + id_col] = m[id_fil*WidthN + id_aux];
			m[id_fil*WidthN + id_aux] = ficha;
		}

		//	Si no hay ningún movimiento de ficha en el hilo, será false
		//	de haberlo, será true
		b[id_fil*WidthN + id_col] = id_col != id_aux;
	}

	//	El resultado de m deberá ser la matriz con las fichas que se pudieran mover a la derecha, movidas,
	//	Y el de b todos los elementos a false, excepto los coincidentes con las fichas que se han podido mover
}

//---------------------- Izq -------------------------

//	Cada hilo busca una pareja para su elemento correspondiente, y si es viable, realiza la suma
//	-	Cada hilo recorre la matriz hacia la izquierda buscando fichas como la suya viables para la suma
//	-	para ello, cuenta cuantas coincidencias hay, si el numero es congruente con 0 mod 2,
//	-	no se realizará ninguna acción por parte del hilo, si es congruente con 1 mod 2,
//	-	se multiplica por 2 el primer coincidente y se borra la ficha del hilo.
//	-	Las coincidencias deben de ser inmediatas, solo permitiendose el 0 entre las fichas (0 == vacio)
//	-	-	La puntuación se recoge en la matriz p
__global__ void SumaIzq(int *m, int *p, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, c = 0, aux, i;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];
		//si la ficha está vacia, el hilo no buscará
		if (ficha != 0) {
			//Se realiza la busqueda hacia la izq
			for (i = id_col - 1; i >= 0; i--) {
				aux = m[id_fil*WidthN + i];

				if (aux == ficha) c++;//contamos las coincidencias
				else if (aux != 0) i = -1;//No podemos emparejar saltandonos fichas
			}

			//	Si el numero de coincidencias es congruente con 1 mod 2
			//	se busca la primera coincidencia, se multiplica por 2 y se borra la ficha 
			//	Si fuese congruente con 0 mod 2, no debe acceder al for
			if ((c % 2) == 0) p[id_fil*WidthN + id_col] = 0;
			for (i = id_col - 1; i >= 0 && (c % 2) == 1; i--) {
				aux = m[id_fil*WidthN + i];
				if (aux == ficha) {
					m[id_fil*WidthN + i] = ficha * 2;
					m[id_fil*WidthN + id_col] = 0;
					p[id_fil*WidthN + id_col] = ficha * 2;//	Grabamos la puntuación obtenido con la suma
					c--;//Para que el bucle for termine
				}

				//	(Aclaración) Si estamos entrando en este bucle for,
				//		significa que se ha encontrado una pareja viable anteriormente
				//		por lo que no se filtra si se opera con una ficha no válida
			}
		}
		else p[id_fil*WidthN + id_col] = 0;
	}
}

//	Ejecución de Movimiento a la Izquierda de las piezas
//	-	Cada hilo toma su ficha (si es distinta de 0) y busca espacios en blanco a su izquierda
//	-	Cuando no encuentra más huecos en la matriz, intercambia su ficha con la del último hueco hallado
//	-	al ser 0, intercambia con una vacía, si no hubiese huecos a su izquierda, la intercambia consigo mismo
//	-	-	Esta función debe ser llamada hasta que no devuelva ningún cambio en la Matriz de Juego
__global__ void exMovIzq(int *m, bool *b, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, id_aux = id_col;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		if (ficha != 0) {//si es 0, no hay que hacer ningún movimiento
			for (int i = id_col - 1; i >= 0; i--) {
				if (m[id_fil*WidthN + i] == 0) id_aux = i;//se va buscando huecos vacios
				else i = -1;//hasta toparse con otra ficha, entonces paramos la búsqueda
			}

			//Intercambiamos las fichas, aunque no se haya encontrado ningún hueco
			m[id_fil*WidthN + id_col] = m[id_fil*WidthN + id_aux];
			m[id_fil*WidthN + id_aux] = ficha;
		}
		

		//	Si no hay ningún movimiento de ficha en el hilo, será false
		//	de haberlo, será true
		b[id_fil*WidthN + id_col] = id_col != id_aux;
	}

	//	El resultado de m deberá ser la matriz con las fichas que se pudieran mover a la izquierda, movidas,
	//	Y el de b todos los elementos a false, excepto los coincidentes con las fichas que se han podido mover
}

//---------------------- Arb -------------------------

//	Cada hilo busca una pareja para su elemento correspondiente, y si es viable, realiza la suma
//	-	Cada hilo recorre la matriz hacia arriba buscando fichas como la suya viables para la suma
//	-	para ello, cuenta cuantas coincidencias hay, si el numero es congruente con 0 mod 2,
//	-	no se realizará ninguna acción por parte del hilo, si es congruente con 1 mod 2,
//	-	se multiplica por 2 el primer coincidente y se borra la ficha del hilo.
//	-	Las coincidencias deben de ser inmediatas, solo permitiendose el 0 entre las fichas (0 == vacio)
//	-	-	La puntuación se recoge en la matriz p
__global__ void SumaArb(int *m, int *p, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, c = 0, aux, i;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		//si la ficha está vacia, el hilo no buscará
		if (ficha != 0) {
			//Se realiza la busqueda hacia arriba
			for (i = id_fil - 1; i >= 0; i--) {
				aux = m[i*WidthN + id_col];

				if (aux == ficha) c++;//contamos las coincidencias
				else if (aux != 0) i = -1;//No podemos emparejar saltandonos fichas
			}

			//	Si el numero de coincidencias es congruente con 1 mod 2
			//	se busca la primera coincidencia, se multiplica por 2 y se borra la ficha 
			//	Si fuese congruente con 0 mod 2, no debe acceder al for
			if ((c % 2) == 0) p[id_fil*WidthN + id_col] = 0;
			for (i = id_fil - 1; i >= 0 && (c % 2) == 1; i--) {
				aux = m[i*WidthN + id_col];
				if (aux == ficha) {
					m[i*WidthN + id_col] = ficha * 2;
					m[id_fil*WidthN + id_col] = 0;
					p[id_fil*WidthN + id_col] = ficha * 2;//	Grabamos la puntuación obtenido con la suma
					c--;//Para que el bucle for termine
				}

				//	(Aclaración) Si estamos entrando en este bucle for,
				//		significa que se ha encontrado una pareja viable anteriormente
				//		por lo que no se filtra si se opera con una ficha no válida
			}
		}
		else p[id_fil*WidthN + id_col] = 0;
	}
}

//	Ejecución de Movimiento hacia Arriba de las piezas
//	-	Cada hilo toma su ficha (si es distinta de 0) y busca espacios en blanco por encima
//	-	Cuando no encuentra más huecos en la matriz, intercambia su ficha con la del último hueco hallado
//	-	al ser 0, intercambia con una vacía, si no hubiese huecos por encima, la intercambia consigo mismo
//	-	-	Esta función debe ser llamada hasta que no devuelva ningún cambio en la Matriz de Juego
__global__ void exMovArb(int *m, bool *b, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, id_aux = id_fil;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		if (ficha != 0) {//si es 0, no hay que hacer ningún movimiento
			for (int i = id_fil - 1; i >= 0; i--) {
				if (m[i*WidthN + id_col] == 0) id_aux = i;//se va buscando huecos vacios
				else i = -1;//hasta toparse con otra ficha, entonces paramos la búsqueda
			}

			//Intercambiamos las fichas, aunque no se haya encontrado ningún hueco
			m[id_fil*WidthN + id_col] = m[id_aux*WidthN + id_col];
			m[id_aux*WidthN + id_col] = ficha;
		}

		//	Si no hay ningún movimiento de ficha en el hilo, será false
		//	de haberlo, será true
		b[id_fil*WidthN + id_col] = id_fil != id_aux;
	}

	//	El resultado de m deberá ser la matriz con las fichas que se pudieran mover hacia arriba, movidas,
	//	Y el de b todos los elementos a false, excepto los coincidentes con las fichas que se han podido mover
}

//---------------------- Abj -------------------------

//	Cada hilo busca una pareja para su elemento correspondiente, y si es viable, realiza la suma
//	-	Cada hilo recorre la matriz hacia abajo buscando fichas como la suya viables para la suma
//	-	para ello, cuenta cuantas coincidencias hay, si el numero es congruente con 0 mod 2,
//	-	no se realizará ninguna acción por parte del hilo, si es congruente con 1 mod 2,
//	-	se multiplica por 2 el primer coincidente y se borra la ficha del hilo.
//	-	Las coincidencias deben de ser inmediatas, solo permitiendose el 0 entre las fichas (0 == vacio)
//	-	-	La puntuación se recoge en la matriz p
__global__ void SumaAbj(int *m, int *p, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, c = 0, aux, i;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		//si la ficha está vacia, el hilo no buscará
		if (ficha != 0) {
			//Se realiza la busqueda hacia abj
			for (i = id_fil + 1; i < WidthM; i++) {
				aux = m[i*WidthN + id_col];

				if (aux == ficha) c++;//contamos las coincidencias
				else if (aux != 0) i = WidthM;//No podemos emparejar saltandonos fichas
			}

			//	Si el numero de coincidencias es congruente con 1 mod 2
			//	se busca la primera coincidencia, se multiplica por 2 y se borra la ficha 
			//	Si fuese congruente con 0 mod 2, no debe acceder al for
			if ((c % 2) == 0)p[id_fil*WidthN + id_col] = 0;
			for (i = id_fil + 1; i < WidthM && (c % 2) == 1; i++) {
				aux = m[i*WidthN + id_col];
				if (aux == ficha) {
					m[i*WidthN + id_col] = ficha * 2;
					m[id_fil*WidthN + id_col] = 0;
					p[id_fil*WidthN + id_col] = ficha * 2;//	Grabamos la puntuación obtenido con la suma
					c--;//Para que el bucle for termine
				}

				//	(Aclaración) Si estamos entrando en este bucle for,
				//		significa que se ha encontrado una pareja viable anteriormente
				//		por lo que no se filtra si se opera con una ficha no válida
			}
		}
		else p[id_fil*WidthN + id_col] = 0;
	}
}

//	Ejecución de Movimiento hacia Abajo de las piezas
//	-	Cada hilo toma su ficha (si es distinta de 0) y busca espacios en blanco por debajo de ella
//	-	Cuando no encuentra más huecos en la matriz, intercambia su ficha con la del último hueco hallado
//	-	al ser 0, intercambia con una vacía, si no hubiese huecos por debajo, la intercambia consigo mismo
//	-	-	Esta función debe ser llamada hasta que no devuelva ningún cambio en la Matriz de Juego
__global__ void exMovAbj(int *m, bool *b, int WidthM, int WidthN) {
	//obtención id del hilo
	/*int idBx = blockIdx.x;	int idBy = blockIdx.y;
	int idTx = threadIdx.x;	int idTy = threadIdx.y;

	int id_fil = idBy * TILE_WIDTH + idTy;//coordenada y
	int id_col = idBx * TILE_WIDTH + idTx;//coordenada x
	*/

	int id_fil = threadIdx.y, id_col = threadIdx.x;

	int ficha, id_aux = id_fil;

	//filtro de hilos
	if (id_fil < WidthM && id_col < WidthN) {
		ficha = m[id_fil*WidthN + id_col];

		if (ficha != 0) {//si es 0, no hay que hacer ningún movimiento
			for (int i = id_fil + 1; i < WidthM; i++) {
				if (m[i*WidthN + id_col] == 0) id_aux = i;//se va buscando huecos vacios
				else i = WidthM;//hasta toparse con otra ficha, entonces paramos la búsqueda
			}

			//Intercambiamos las fichas, aunque no se haya encontrado ningún hueco
			m[id_fil*WidthN + id_col] = m[id_aux*WidthN + id_col];
			m[id_aux*WidthN + id_col] = ficha;
		}

		//	Si no hay ningún movimiento de ficha en el hilo, será false
		//	de haberlo, será true
		b[id_fil*WidthN + id_col] = id_fil != id_aux;
	}

	//	El resultado de m deberá ser la matriz con las fichas que se pudieran mover hacia abajo, movidas,
	//	Y el de b todos los elementos a false, excepto los coincidentes con las fichas que se han podido mover
}

//---------------------------------------------------------------------------------------------------------------------------------------------------

//-------------------------------------------------------------------- Host -------------------------------------------------------------------------

enum Colores {	//Colores para el fondo y la fuente de la consola
	BLACK = 0,
	BLUE = 1,
	GREEN = 2,
	CYAN = 3,
	RED = 4,
	MAGENTA = 5,
	BROWN = 6,
	LGREY = 7,
	DGREY = 8,
	LBLUE = 9,
	LGREEN = 10,
	LCYAN = 11,
	LRED = 12,
	LMAGENTA = 13,
	YELLOW = 14,
	WHITE = 15
};

//	Cambia el color de fondo y de fuente de la consola
void Color(int fondo, int fuente) {

	HANDLE Consola = GetStdHandle(STD_OUTPUT_HANDLE);
	//Cálculo para convertir los colores al valor necesario
	int color_nuevo = fuente + (fondo * 16);
	//Aplicamos el color a la consola
	SetConsoleTextAttribute(Consola, color_nuevo);

}

//	Inicializador de la matriz de juego
//	-	*m Matriz en forma vectorial con la que se trabaja, WidthM y WidthN su tamaño de columna y fila
//	-	x e y, las coordenadas del elemento que se introducira con el valor indicado
bool IntroCasilla(int *m, int WidthN, int x, int y, int valor) {
	bool out = m[y*WidthN + x] == 0;

	if (out) m[y*WidthN + x] = valor;

	return out;
}

void obtenerCaracteristicas(int n_columnas, int n_filas) {
	hipDeviceProp_t prop;
	hipGetDeviceProperties(&prop, 0);
	int size = n_columnas*n_filas;//Numero de elementos de los tableros de juego
	
	//printf("Características de la tarjeta: \n");
	//printf("Nombre: %s \n", prop.name);
	//printf("Capabilities: %d.%d \n", prop.major, prop.minor);
	//printf("Maximo de hilos por bloque: %d \n", prop.maxThreadsPerBlock);
	//printf("Maximo de hilos por SM: %d \n", prop.maxThreadsPerMultiProcessor);
	//printf("Maximo de memoria global: %zd \n", prop.totalGlobalMem);
	//printf("Maximo de memoria compartida: %zd \n", prop.sharedMemPerBlock);
	//printf("Maximo de registros: %d \n", prop.regsPerBlock);
	//printf("Numero de multiprocesadores: %d \n", prop.multiProcessorCount);

	//	Tamaño de la matriz en hilos y memoria
	//printf("Numero de hilos de la matriz: %d \n", n_columnas*n_filas);
	//printf("Cantidad de memoria utilizada por la matriz: %zd \n", n_columnas*n_filas * sizeof(int));

	//	El máximo de hilos necesario es igual al numero de elementos de la matriz
	if (prop.maxThreadsPerBlock < (size)) {
		printf("No hay suficientes hilos disponibles para calcular la matriz\n");
		exit(-1);
	}

	//	El tamaño de memoria máximo necesario es equivalente a almacenar dos tableros,
	//		uno de int y otro de bool
	if (prop.totalGlobalMem < ((size * sizeof(int)) + (size * sizeof(bool)))) {
		printf("El tamaño de memoria global es insuficiente para calcular la matriz \n");
		exit(-1);
	}
}

//	Carga los datos del Juego Guardados anteriormente
//	-	v Matriz de Juego, WidthM y WidthN el tamaó de la matriz de juego,
//	-	puntuacion de juego acumulada y s el nombre del archivo de guardado
bool cargaDatos(int **v, int *WidthM, int *WidthN, int *puntuacion, int *vidas, int *dificultad, char *s) {
	string linea; //buffer de entrada
	char *c; //cadena de char para transformación
	char *token; //tokens de la matriz
	int *m; //Matriz de entrada
	int i, k; //contador de acceso a la matriz y longitud de esta
	ifstream entrada(s, ios::binary); //Fichero de entrada
	bool out = entrada.is_open(); //Si ha logrado abrir el archivo y no ha resultado erroneo la carga de datos

	if (out) {
		//	Carga del numero de filas de la matriz de juego
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size()+1) * sizeof(char));
			strcpy(c, linea.c_str());
			*WidthM = atoi(c);
			free((void *)c);
		}
		else out = false;
		//	Carga del numero de columnas de la matriz de juego
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());
			*WidthN = atoi(c);
			free((void *)c);
		}
		else out = false;
		//	Carga de la puntuación
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());
			*puntuacion = atoi(c);
			free((void *)c);
		}
		else out = false;
		//	Carga el numero de vidas
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());
			*vidas = atoi(c);
			free((void *)c);
		}
		else out = false;
		//	Carga la dificultad
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());
			*dificultad = atoi(c);
			free((void *)c);
		}
		else out = false;
		//	Carga el modo
		if (getline(entrada, linea, ';')) {
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());
			MODO[0] = c[0];
			MODO[1] = c[1];
			free((void *)c);
		}
		else out = false;
		//	Carga de la matriz de juego
		if (getline(entrada, linea, ';')) {//	Toma hasta el ';'
			c = (char*)malloc((linea.size() + 1) * sizeof(char));
			strcpy(c, linea.c_str());//	Pasamos a char para trabajar con ello
			token = strtok(c, " ,");//	Extraemos los numeros con Tokens
			k = *WidthM * *WidthN;	//	longitud de v
			m = (int *)malloc(k * sizeof(int));
			*v = m;
			i = 0;

			while (token != NULL) {
				m[i] = atoi(token);//	Introducimos el cada numero en la matriz obteniendo Tokens
				token = strtok(NULL, " ,");
				i++;
			}

			free((void *)c);//	liberamos la memoria reservada para el char *
		}
		else out = false;

		entrada.close();//Cerramos el archivo
		if (!out) { fprintf(stderr, "Fallo al cargar los datos de guardado\n");}
	}
	else { fprintf(stderr, "Fallo al intentar abrir el archivo de guardado\n");}
	
	return out;
}

//	Guarda los datos de juego en el archivo de guardado indicado
//	-	v matriz de juego, WidthM y WidthN dimensiones de la matriz de juego,
//	-	puntuación de la partida y nombre del archivo destino
void guardaDatos(int *v, int WidthM, int WidthN, int puntuacion, int vidas, int dificultad, char *s) {
	ofstream salida;//	Fichero de salida
	int Width = WidthM * WidthN;//	Numero de elementos de la matriz de juego
	salida.open(s);//	Abrimos el fichero
	if (salida.is_open()) {
		//	El resto de los datos los grabamos separandolos con ';'
		salida << WidthM << ";" << WidthN << ";" << puntuacion << ";" << vidas << ";" << dificultad << ";" << MODO << ";";
		for (int i = 0; i < Width; i++) {	//	Recorremos la matriz y
			salida << v[i];					//	vamos grabando los elementos en el archivo
			if (i < Width - 1) salida << ",";//	separando con ','
		}
		salida << ";";

		salida.close();//	Cerramos el archivo
	}
	else fprintf(stderr, "Fallo al intentar abrir el archivo de guardado\n");
}

//	Leemos de teclado y devolvemos un numero de salida en funcion de la tecla pulsada
int reconocerTeclado() {
	char tecla;
	int salida;
	bool flag = true;

	do {
		tecla = getch();

		if (tecla == 'p' || tecla == 'P') { salida = 0; flag = false;}
		if (tecla == 'w' || tecla == 'W') { salida = 1; flag = false;}
		if (tecla == 'a' || tecla == 'A') { salida = 2; flag = false;}
		if (tecla == 'd' || tecla == 'D') {salida = 3; flag = false;}
		if (tecla == 's' || tecla == 'S') {salida = 4; flag = false;}

		if (tecla == 'r' || tecla == 'R') {salida = 5; flag = false;}
		if (tecla == 'g' || tecla == 'G') {salida = 6; flag = false;}

		if (tecla == -32) {
			tecla = getch();
			if (tecla == 72) {salida = 1; flag = false;}
			if (tecla == 75) {salida = 2; flag = false;}
			if (tecla == 77) {salida = 3; flag = false;}
			if (tecla == 80) {salida = 4; flag = false;}
		}

		if (tecla == 13) {salida = 7; flag = false;}
	} while (flag);

	return salida;
}

//	Mostramos una introduccion
void mostrarMenuInicial() {

	printf(".----------------.  .----------------.  .----------------.  .----------------.  .----------------.\n");
	printf("| .--------------. || .--------------. || .--------------. || .--------------. || .--------------. |\n");
	printf("| |     __       | || |    ______    | || |    ______    | || |     ____     | || |   _    _     | |\n");
	printf("| |    /  |      | || |  .' ____ \\   | || |   / ____ `.  | || |   .' __ '.   | || |  | |  | |    | |\n");
	printf("| |    `| |      | || |  | |____\\_|  | || |   `'  __) |  | || |   | (__) |   | || |  | |__| |_   | |\n");
	printf("| |     | |      | || |  | '____`'.  | || |   _ | __ '.  | || |   .`____'.   | || |  |____   _|  | |\n");
	printf("| |    _| |_     | || |  | (____) |  | || |  | \\____) |  | || |  | (____) |  | || |      _| |_   | |\n");
	printf("| |   |_____|    | || |  '.______.'  | || |   \\______.'  | || |  `.______.'  | || |     |_____|  | |\n");
	printf("| |              | || |              | || |              | || |              | || |              | |\n");
	printf("| '--------------' || '--------------' || '--------------' || '--------------' || '--------------' |\n");
	printf("'----------------'  '----------------'  '----------------'  '----------------'  '----------------' \n\n");
	printf("                       Created by: Diego-Edgar Gracia & Daniel Lopez                                \n\n");
	printf("                                                                                                      \n\n");

}

//	Mostramos menu de carga de datos
int menuCargaDatos() {
	bool flag = true;
	int sel = 2;//	0 Cargar	/	1 Nueva	/	2 Salir

	do {
		system("cls");
		Color(WHITE, BLACK);
		printf("Se ha encontrado una partida anterior\n%cDeseas cargar esa partida?          \n\n",168);
		switch (sel)
		{
		case 0: Color(LCYAN, BLACK); printf("- Cargar Partida\n");
			Color(BLACK, WHITE); printf("- Nueva Partida\n- Salir\n");
			break;
		case 1: Color(BLACK, WHITE); printf("- Cargar Partida\n");
			Color(LCYAN, BLACK); printf("- Nueva Partida\n");
			Color(BLACK, WHITE); printf("- Salir\n");
			break;
		case 2: Color(BLACK, WHITE); printf("- Cargar Partida\n- Nueva Partida\n");
			Color(LCYAN, BLACK); printf("- Salir\n"); Color(BLACK, WHITE);
			break;
		default:
			break;
		}

		switch (reconocerTeclado())
		{
		case 1: sel--;
			if (sel < 0) sel = 2;
			break;
		case 4: sel++;
			if (sel > 2) sel = 0;
			break;
		case 7: flag = false;
			break;
		default:
			break;
		}
	} while (flag);

	return sel;
}

//	Mostramos las opciones de pausa
void mostrarMenuPausa() {
	system("cls");
	Color(WHITE, BLACK);
	printf("                        PAUSA             ");
	Color(BLACK, WHITE);
	printf("\n\n");
	printf("Selecciona una opcion:\n");
	printf("\t R - Reanudar \n");
	printf("\t G - Guardar progreso y salir \n");
	printf("\t S - Salir sin guardar \n");
}

//	Imprime la matriz de juego
//	-	Recorre las filas de la matriz de juego
void imprimeMatriz(int *p, int *v, int m, int n) {//( m * n )
	int i, j, x;
	int ws;//numero de espacios de caracteres por casilla
	printf("\n");
	system("cls");
	Color(WHITE, BLACK);
	printf("-WASD y Flechas del Teclado para mover las fichas\n-P para Pausa                                    \n");
	Color(BLACK, WHITE);
	printf("Puntuacion: %d \n", *p);
	for (i = 0; i < m; i++) {//recorremos eje m
		for (j = 0; j < n; j++) {//recorremos eje n
			ws = WS;
			x = v[i*n + j];

			//No se consideran numeros negativos, y el límite son 6 dígitos (que no se alcanzan)

			do {//Se ocupa un hueco por digito del numero
				ws--;
				x = x / 10;
			} while (x > 0);

			switch (v[i*n + j]) {//	Modifica el color en el que se mostrarán los elementos
			case 0:
				Color(BLACK, RED);
				break;
			case 2:
				Color(WHITE, BLACK);
				break;
			case 4:
				Color(YELLOW, BLACK);
				break;
			case 8:
				Color(LMAGENTA, BLACK);
				break;
			case 16:
				Color(MAGENTA, BLACK);
				break;
			case 32:
				Color(BROWN, BLACK);
				break;
			case 64:
				Color(RED, BLACK);
				break;
			case 128:
				Color(LBLUE, BLACK);
				break;
			case 256:
				Color(BLUE, BLACK);
				break;
			case 512:
				Color(LGREEN, BLACK);
				break;
			case 1024:
				Color(GREEN, BLACK);
				break;
			case 2048:
				Color(LGREY, BLACK);
				break;
			case 4096:
				Color(DGREY, BLACK);
				break;
			case 8192:
				Color(CYAN, BLACK);
				break;
			case 16384:
				Color(WHITE, BLACK);
				break;
			default:
				Color(BLACK, WHITE);
				break;
			}

			printf("%d", v[i*n + j]);//imprimimos el numero
			while (ws > 0) {//y ocupamos el resto de huecos con espacios en blanco
				if (ws == 1) {
					Color(BLACK, WHITE);
				}
				printf(" ");
				ws--;
			}
		}
		printf("\n");
		
	}
	printf("\n");

	if (VIDAS > 0) {
		Color(WHITE, BLACK);
		printf("VIDAS:              ");
		Color(BLACK, WHITE);
		printf("\n");
		Color(WHITE, RED);
		for (int i = 0; i < VIDAS; i++) {
			printf(" <3 ");
		}
		Color(WHITE, BLACK);
		printf("\n");
	}
	

	Color(BLACK, WHITE);
}

//Solo para pruebas
/*
void imprimeBooleanos(bool *v, int m, int n) {//( m * n )
int i, j;
bool x;
int ws;//numero de espacios de caracteres por casilla
printf("\n");
for (i = 0; i < m; i++) {//recorremos eje m
for (j = 0; j < n; j++) {//recorremos eje n
ws = WS;
x = v[i*n + j];
if (v[i*n + j]) { printf("True"); ws = ws - 4; }
else { printf("False"); ws = ws - 5; }
while (ws > 0) {//y ocupamos el resto de huecos con espacios en blanco
printf(" ");
ws--;
}
}
printf("\n");
}
}*/

//	Introduce en la matriz de juego un nuevo numero
//	-	*m matriz de Juego, WidthM y WidthN dimensiones de columna y fila
//	-	x e y, coordenadas donde se intenta introducir el elemento "set", si ya hay un elemento (!= 0), no se introduce y devuelve false
bool introNum(int *m, int WidthM, int WidthN, int x, int y, int set) {
	//comprobación de que esté dentro
	if (x < WidthN && y < WidthM) {
		if (m[y*WidthN + x] == 0) {
			m[y*WidthN + x] = set;
			return true;
		}
	}

	return false;
}

//	Comprueba si hay al menos un elemento verdadero en la matriz
bool checkMatrizBool(bool *b, int m, int n) {
	bool out = false;
	int i, j;

	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			out = out || b[i*n + j];
		}
	}

	return out;
}

//	Comprueba si hay al menos una casilla vacia
bool checkLleno(int *v, int m, int n) {
	bool out = false;
	int i, j;

	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			out = out || (v[i*n + j] == 0);
		}
	}

	return out;
}

//	Suma la puntuación total de la matriz
//	-	Recorremos toda la matriz buscando la puntuación total que guarda
int sumaPuntuacion(int *p, int m, int n) {
	int out = 0;
	int i, j;

	for (i = 0; i < m; i++) {
		for (j = 0; j < n; j++) {
			out = out + p[i*n + j];
		}
	}

	return out;
}

void introSemilla(int *v, int WidthM, int WidthN, int dificultad) {
	int x, y, valor;

	//Metemos nuevas semillas despues de realizar cada movimiento
	if (dificultad == 1) {
		for (int i = 0; i < 15; i++) {
			do {
				x = rand() % WidthN;
				y = rand() % WidthM;
				valor = BAJO[rand() % 3];
			} while (!IntroCasilla(v, WidthN, x, y, valor) && checkLleno(v, WidthM, WidthN));
		}
	}
	else if (dificultad == 2) {
		for (int i = 0; i < 8; i++) {
			do {
				x = rand() % WidthN;
				y = rand() % WidthM;
				valor = BAJO[rand() % 2];
			} while (!IntroCasilla(v, WidthN, x, y, valor) && checkLleno(v, WidthM, WidthN));
		}
	}
}

//	Realizamos las sumas y los movimientos hacia arriba
//	-	v Matriz de juego, p UN SOLO ENTERO CON LA PUNTUACIÓN,
//	-	la propia función se encargará de obtenerlo
hipError_t accionArriba(int *v, int *p, int WidthM, int WidthN) {
	//printf("accion arriba");
	hipError_t cudaStatus;
	int *dev_v = 0, *dev_p = 0;
	bool *dev_b = 0;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(WidthN, WidthM);

	int *h_p = (int*) malloc(WidthM * WidthN * sizeof(int));
	bool *b = (bool*) malloc(WidthM * WidthN * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto FreeArb;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_v, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeArb;
	}

	cudaStatus = hipMalloc((void**)&dev_p, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeArb;
	}

	cudaStatus = hipMemcpy(dev_v, v, WidthM * WidthN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeArb;
	}

	//	Sumamos las fichas que se puedan juntar

	//printf("Sumamos");
	SumaArb << <dimGrid, dimBlock >> >(dev_v, dev_p, WidthM, WidthN);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "SumaArb launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto FreeArb;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SumaArb!\n", cudaStatus);
		goto FreeArb;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeArb;
	}

	cudaStatus = hipMemcpy(h_p, dev_p, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeArb;
	}

	//printf(" sumando puntuacion ");
	*p = *p + sumaPuntuacion(h_p, WidthM, WidthN);

	cudaStatus = hipMalloc((void**)&dev_b, WidthM * WidthN * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeArb;
	}

	do {
		//printf("\nMovemos casillas ");
		//	Inicializamos la matriz de bool

		//	Rellena de False la matriz de booleanos
		//printf("bools");
		iniBool << <dimGrid, dimBlock >> > (dev_b, WidthM, WidthN, false);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "iniBool launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeArb;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching iniBool!\n", cudaStatus);
			goto FreeArb;
		}

		//	Movemos

		exMovArb << <dimGrid, dimBlock >> > (dev_v, dev_b, WidthM, WidthN);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "exMovArb launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeArb;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching esMovArb!\n", cudaStatus);
			goto FreeArb;
		}

		cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeArb;
		}

		cudaStatus = hipMemcpy(b, dev_b, WidthM * WidthN * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeArb;
		}

		//Mientras se haya movido una ficha se vuelve a ejecutar el movimiento
	} while (checkMatrizBool(b, WidthM, WidthN));

	//printf("\nTodo Movido");

FreeArb:
	hipFree(dev_v);
	hipFree(dev_p);
	hipFree(dev_b);
	free(h_p);
	free(b);

	return cudaStatus;
}

//Realizamos las sumas y los movimientos hacia la izquierda
hipError_t accionIzquierda(int *v, int *p, int WidthM, int WidthN) {
	//printf("accion izquierda");
	hipError_t cudaStatus;
	int *dev_v = 0, *dev_p = 0;
	bool *dev_b = 0;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(WidthN, WidthM);

	int *h_p = (int*)malloc(WidthM * WidthN * sizeof(int));
	bool *b = (bool*)malloc(WidthM * WidthN * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto FreeIzq;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_v, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeIzq;
	}

	cudaStatus = hipMalloc((void**)&dev_p, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeIzq;
	}

	cudaStatus = hipMemcpy(dev_v, v, WidthM * WidthN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeIzq;
	}

	//	Sumamos las fichas que se puedan juntar

	//printf("Sumamos");
	SumaIzq << <dimGrid, dimBlock >> > (dev_v, dev_p, WidthM, WidthN);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "SumaIzq launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto FreeIzq;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SumaIzq!\n", cudaStatus);
		goto FreeIzq;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeIzq;
	}

	cudaStatus = hipMemcpy(h_p, dev_p, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeIzq;
	}

	//printf(" sumando puntuacion ");
	*p = *p + sumaPuntuacion(h_p, WidthM, WidthN);

	cudaStatus = hipMalloc((void**)&dev_b, WidthM * WidthN * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeIzq;
	}

	do {
		//printf("\nMovemos casillas ");
		//	Inicializamos la matriz de bool

		//	Rellena de False la matriz de booleanos
		//printf("bools");
		iniBool << <dimGrid, dimBlock >> > (dev_b, WidthM, WidthN, false);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "iniBool launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeIzq;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching iniBool!\n", cudaStatus);
			goto FreeIzq;
		}

		//	Movemos

		exMovIzq << <dimGrid, dimBlock >> > (dev_v, dev_b, WidthM, WidthN);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "exMovIzq launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeIzq;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching exMovIzq!\n", cudaStatus);
			goto FreeIzq;
		}

		cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeIzq;
		}

		cudaStatus = hipMemcpy(b, dev_b, WidthM * WidthN * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeIzq;
		}

		//Mientras se haya movido una ficha se vuelve a ejecutar el movimiento
	} while (checkMatrizBool(b, WidthM, WidthN));

	//printf("\nTodo Movido");

FreeIzq:
	hipFree(dev_v);
	hipFree(dev_p);
	hipFree(dev_b);
	free(h_p);
	free(b);

	return cudaStatus;
}

//Realizamos las sumas y los movimientos hacia la derecha
hipError_t accionDerecha(int *v, int *p, int WidthM, int WidthN) {
	//printf("accion derecha");
	hipError_t cudaStatus;
	int *dev_v = 0, *dev_p = 0;
	bool *dev_b = 0;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(WidthN, WidthM);

	int *h_p = (int*)malloc(WidthM * WidthN * sizeof(int));
	bool *b = (bool*)malloc(WidthM * WidthN * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto FreeDch;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_v, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeDch;
	}

	cudaStatus = hipMalloc((void**)&dev_p, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeDch;
	}

	cudaStatus = hipMemcpy(dev_v, v, WidthM * WidthN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeDch;
	}

	//	Sumamos las fichas que se puedan juntar

	//printf("Sumamos");
	SumaDch << <dimGrid, dimBlock >> > (dev_v, dev_p, WidthM, WidthN);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "SumaDch launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto FreeDch;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SumaDch!\n", cudaStatus);
		goto FreeDch;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeDch;
	}

	cudaStatus = hipMemcpy(h_p, dev_p, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeDch;
	}

	//printf(" sumando puntuacion ");
	*p = *p + sumaPuntuacion(h_p, WidthM, WidthN);

	cudaStatus = hipMalloc((void**)&dev_b, WidthM * WidthN * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeDch;
	}

	do {
		//printf("\nMovemos casillas ");
		//	Inicializamos la matriz de bool

		//	Rellena de False la matriz de booleanos
		//printf("bools");
		iniBool << <dimGrid, dimBlock >> > (dev_b, WidthM, WidthN, false);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "iniBool launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeDch;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching iniBool!\n", cudaStatus);
			goto FreeDch;
		}

		//	Movemos

		exMovDch << <dimGrid, dimBlock >> > (dev_v, dev_b, WidthM, WidthN);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "exMovDch launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeDch;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching exMovDch!\n", cudaStatus);
			goto FreeDch;
		}

		cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeDch;
		}

		cudaStatus = hipMemcpy(b, dev_b, WidthM * WidthN * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeDch;
		}

		//Mientras se haya movido una ficha se vuelve a ejecutar el movimiento
	} while (checkMatrizBool(b, WidthM, WidthN));

	//printf("\nTodo Movido");

FreeDch:
	hipFree(dev_v);
	hipFree(dev_p);
	hipFree(dev_b);
	free(h_p);
	free(b);

	return cudaStatus;
}

//Realizamos las sumas y los movimientos hacia abajo
hipError_t accionAbajo(int *v, int *p, int WidthM, int WidthN) {
	//printf("accion abajo");
	hipError_t cudaStatus;
	int *dev_v = 0, *dev_p = 0;
	bool *dev_b = 0;
	dim3 dimGrid(1, 1);
	dim3 dimBlock(WidthN, WidthM);

	int *h_p = (int*) malloc(WidthM * WidthN * sizeof(int));
	bool *b = (bool*) malloc(WidthM * WidthN * sizeof(bool));

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto FreeAbj;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_v, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeAbj;
	}

	cudaStatus = hipMalloc((void**)&dev_p, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeAbj;
	}

	cudaStatus = hipMemcpy(dev_v, v, WidthM * WidthN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeAbj;
	}

	//	Sumamos las fichas que se puedan juntar
	//printf("Sumamos");
	SumaAbj << <dimGrid, dimBlock >> > (dev_v, dev_p, WidthM, WidthN);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "SumaAbj launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto FreeAbj;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching SumaAbj!\n", cudaStatus);
		goto FreeAbj;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeAbj;
	}

	cudaStatus = hipMemcpy(h_p, dev_p, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeAbj;
	}

	//printf(" sumando puntuacion ");
	*p = *p + sumaPuntuacion(h_p, WidthM, WidthN);

	cudaStatus = hipMalloc((void**)&dev_b, WidthM * WidthN * sizeof(bool));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto FreeAbj;
	}

	do {
		//printf("\nMovemos casillas ");
		//	Inicializamos la matriz de bool

		//	Rellena de False la matriz de booleanos
		//printf("bools");
		iniBool << <dimGrid, dimBlock >> > (dev_b, WidthM, WidthN, false);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "iniBool launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeAbj;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching iniBool!\n", cudaStatus);
			goto FreeAbj;
		}

		//	Movemos

		exMovAbj << <dimGrid, dimBlock >> > (dev_v, dev_b, WidthM, WidthN);

		cudaStatus = hipGetLastError();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "exMovAbj launch failed: %s\n", hipGetErrorString(cudaStatus));
			goto FreeAbj;
		}

		cudaStatus = hipDeviceSynchronize();
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching exMovAbj!\n", cudaStatus);
			goto FreeAbj;
		}

		cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeAbj;
		}

		cudaStatus = hipMemcpy(b, dev_b, WidthM * WidthN * sizeof(bool), hipMemcpyDeviceToHost);
		if (cudaStatus != hipSuccess) {
			fprintf(stderr, "hipMemcpy failed!");
			goto FreeAbj;
		}

		//Mientras se haya movido una ficha se vuelve a ejecutar el movimiento
	} while (checkMatrizBool(b, WidthM, WidthN));

	//printf("\nTodo Movido");

FreeAbj:
	hipFree(dev_v);
	hipFree(dev_p);
	hipFree(dev_b);
	free(h_p);
	free(b);

	return cudaStatus;
}

//Salimos del juego sin guardar partida
void accionSalir() {
	printf("Salir \n");
	exit(-1);
}

//Volvemos al juego
void accionReanudar() {
	printf("Reanudar \n");
}

//Guardamos el progreso y salimos
void accionGuardarSalir(int *v, int WidthM, int WidthN, int puntuacion, int dificultad) {
	printf("Guardar y salir \n");
	guardaDatos(v, WidthM, WidthN, puntuacion, VIDAS, dificultad, FICHERO);
	exit(-1);
}

//	Ejecutamos una accion en funcion de la tecla pulsada
void accionPausa(int *v, int WidthM, int WidthN, int puntuacion, int dificultad) {
	int tecla;
	bool flag = true;

	do {
		tecla = reconocerTeclado();

		switch (tecla) {
			//Salir sin guardar
			case 4:
				accionSalir();
				break;
			//Reanudar
			case 5:
				accionReanudar();
				flag = false;
				break;
			//Guardar y salir
			case 6:
				accionGuardarSalir(v, WidthM, WidthN, puntuacion, dificultad);
				break;
		}
	} while (flag);
}

//	IA de juego automático por busqueda de la mejor puntuación en cada iteración
void modoAutomatico(int **v, int dificultad, int *puntuacion, int WidthM, int WidthN) {
	size_t sizeV = WidthM * WidthN * sizeof(int);

	//Puntuaciones
	int *p = (int*)malloc(sizeof(int));
	int *p1 = (int*)malloc(sizeof(int));
	int *p2 = (int*)malloc(sizeof(int));
	int *p3 = (int*)malloc(sizeof(int));
	int *p4 = (int*)malloc(sizeof(int));

	*p = *puntuacion;
	*p1 = *p;
	*p2 = *p;
	*p3 = *p;
	*p4 = *p;

	//Tableros de las distintas opciones de juego
	int *v1 = (int *) malloc(sizeV);
	int *v2 = (int *) malloc(sizeV);
	int *v3 = (int *) malloc(sizeV);
	int *v4 = (int *) malloc(sizeV);

	imprimeMatriz(p, *v, WidthM, WidthN);
	//getch();
	for (int i = 5; i > 0; i--) { printf("%d\n", i); Sleep(1000); }

	do {//Juega

		//Copia el tablero actual
		memcpy(v1, *v, sizeV);
		memcpy(v2, *v, sizeV);
		memcpy(v3, *v, sizeV);
		memcpy(v4, *v, sizeV);
		//Ejecuta los movimientos en los 4 tableros alternos
		accionArriba(v1, p1, WidthM, WidthN);
		accionIzquierda(v2, p2, WidthM, WidthN);
		accionDerecha(v3, p3, WidthM, WidthN);
		accionAbajo(v4, p4, WidthM, WidthN);
		//Busca cual da la mejor puntuación
		memcpy(*v, v1, sizeV); *p = *p1;
		if (*p < *p2) { *p = *p2; memcpy(*v, v2, sizeV); }
		if (*p < *p3) { *p = *p3; memcpy(*v, v3, sizeV); }
		if (*p < *p4) { *p = *p4; memcpy(*v, v4, sizeV); }
		//lo muestra
		imprimeMatriz(p, *v, WidthM, WidthN);
		//getch();
		Sleep(500);
		//Introduce nuevas casillas
		introSemilla(*v, WidthM, WidthN, dificultad);
		imprimeMatriz(p, *v, WidthM, WidthN);
		//getch();
		Sleep(500);
		//Y vuelta a empezar hasta que no pueda continuar
	} while (checkLleno(*v, WidthM, WidthN));

	free((void *) p);
	free((void *) p1);
	free((void *) p2);
	free((void *) p3);
	free((void *) p4);
	free((void *) v1);
	free((void *) v2);
	free((void *) v3);
	free((void *) v4);
}

void modoManual(int *v, int dificultad, int *puntuacion, int WidthM, int WidthN) {
	int teclaPulsada;
	int *p = (int*) malloc(sizeof(int));

	*p = *puntuacion;

	do {

		imprimeMatriz(p, v, WidthM, WidthN);

		do { teclaPulsada = reconocerTeclado(); } while (teclaPulsada>4);
		//printf("%d", teclaPulsada);
		switch (teclaPulsada) {
			//Menu de Pausa
		case 0:
			mostrarMenuPausa();
			accionPausa(v, WidthM, WidthN, *p, dificultad);
			break;

			//Arriba
		case 1:
			accionArriba(v, p, WidthM, WidthN);
			introSemilla(v, WidthM, WidthN, dificultad);
			break;

			//Izquierda
		case 2:
			accionIzquierda(v, p, WidthM, WidthN);
			introSemilla(v, WidthM, WidthN, dificultad);
			break;

			//Derecha
		case 3:
			accionDerecha(v, p, WidthM, WidthN);
			introSemilla(v, WidthM, WidthN, dificultad);
			break;

			//Abajo
		case 4:
			accionAbajo(v, p, WidthM, WidthN);
			introSemilla(v, WidthM, WidthN, dificultad);
			break;

			//Defecto
		default:
			break;
		}

		//imprimeMatriz(p, v, WidthM, WidthN);

		//printf("\n - fin loop - ");
	} while (checkLleno(v, WidthM, WidthN));

	//printf("\nMatriz llena - fin de partida");

	if (!checkLleno(v, WidthM, WidthN)) {
		VIDAS--;
	}
	

//Morgan
//FreeMan:
	free((void *) v);
	free((void *) p);
}

hipError_t iniciaMatriz(int *v, int WidthM, int WidthN, int dificultad) {
	int *dev_v = 0;

	dim3 dimGrid(1, 1);
	dim3 dimBlock(WidthN, WidthM);

	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto FreeIni;
	}

	// Allocate GPU buffers
	cudaStatus = hipMalloc((void**)&dev_v, WidthM * WidthN * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc iniciaMatriz failed!");
		goto FreeIni;
	}

	cudaStatus = hipMemcpy(dev_v, v, WidthM * WidthN * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeIni;
	}

	Inicializador << <dimGrid, dimBlock >> > (dev_v, WidthM, WidthN);

	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "Inicializador launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto FreeIni;
	}
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching Inicializador!\n", cudaStatus);
		goto FreeIni;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(v, dev_v, WidthM * WidthN * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto FreeIni;
	}

	introSemilla(v, WidthM, WidthN, dificultad);

FreeIni:
	hipFree((void *) dev_v);

	return cudaStatus;
}

int main(int argc, char** argv) {
	//Establecemos la semilla del random
	srand(time(NULL));

	//Mostramos el menú inicial y procedemos a jugar
	mostrarMenuInicial();

	int *v = 0, WidthM, WidthN, *punt = (int *) malloc(sizeof(int)), vidas, dificultad;
	*punt = 0;
	int sel = -1;

	if (argc < 5) { 
		if (cargaDatos(&v, &WidthM, &WidthN, punt, &vidas, &dificultad, FICHERO)) printf("\nDatos anteriores Cargados\n");
		else { fprintf(stderr, "\nNo se han introducido detalles de partida ni hay un archivo de guardado previo\n"); exit(-1); }
	} else {
		if (cargaDatos(&v, &WidthM, &WidthN, punt, &vidas, &dificultad, FICHERO)) {

			sel = menuCargaDatos();

			switch (sel) {
			case 0:
				//	Antes de ejecutar nada en la GPU, comprobamos que se pueda ejecutar
				obtenerCaracteristicas(WidthM, WidthN);
				break;
			case 1:
				WidthM = atoi(argv[3]);
				WidthN = atoi(argv[4]);
				VIDAS = 5;
				dificultad = atoi(argv[2]);
				strcpy(MODO, argv[1]);
				v = (int*) malloc(WidthM * WidthN * sizeof(int));

				//	Antes de ejecutar nada en la GPU, comprobamos que se pueda ejecutar
				obtenerCaracteristicas(WidthM, WidthN);

				iniciaMatriz(v, WidthM, WidthN, dificultad);
				break;
			case 2:
				printf("Bye Bye!\n");
				exit(0);
				break;
			default:
				fprintf(stderr, "%cComo has llegado aquí?\n", 168);
				exit(-1);
				break;
			}
		}
		else {
			WidthM = atoi(argv[3]);
			WidthN = atoi(argv[4]);
			VIDAS = 5;
			dificultad = atoi(argv[2]);
			strcpy(MODO, argv[1]);
			v = (int*)malloc(WidthM*WidthN * sizeof(int));

			//	Antes de ejecutar nada en la GPU, comprobamos que se pueda ejecutar
			obtenerCaracteristicas(WidthM, WidthN);

			iniciaMatriz(v, WidthM, WidthN, dificultad);
		}
	}



	//Modo Automatico
	if (strcmp(MODO, "-a") == 0) {
		VIDAS = 0;//La IA no necesita VIDAS (es así de chulita)
		modoAutomatico(&v, dificultad, punt, WidthM, WidthN);
	}

	//Modo Manual
	else if (strcmp(MODO, "-m") == 0) {
		do {
			modoManual(v, dificultad, punt, WidthM, WidthN);
			if (VIDAS > 0) {
				*punt = 0;
				v = (int*) malloc(WidthM * WidthN * sizeof(int));
				iniciaMatriz(v, WidthM, WidthN, dificultad);
			}
		} while (VIDAS>0);
	}

	free((void *) v);
	free((void *) punt);
}

//---------------------------------------------------------------------------------------------------------------------------------------------------
